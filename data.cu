#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>

#include "vtk.h"
#include "data.h"

struct Constants constants = {
	.c = 299792458,
	.mu = 4.0 * M_PI * 1.0e-7,
	.eps = 1.0 / (constants.c * constants.c * constants.mu),
	.cfl = 0.6363961031
};

struct Specifics specifics;
struct Arrays arrays;
struct Cuda cuda_consts;

double *** host_E;
double *** host_B;

// Time to run for / or number of steps
double T = 0.0001;
int steps = 0;

/**
 * @brief Allocate a 2D CUDA array that is addressable using square brackets
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @return double** A 2D array
 */
void alloc_2d_cuda_array(int m, int n, double **array, size_t *pitch) {
  	hipMallocPitch((void **)array, pitch, n*sizeof(double), m);
	*pitch = (*pitch) / sizeof(double);
	// double *tmp = (double *) calloc((*pitch)*m, sizeof(double));
	// hipMemcpy(*array, tmp, (*pitch)*m*sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Free a 2D CUDA array
 * 
 * @param array The 2D array to free
 */
void free_2d_cuda_array(double *array) {
	hipFree(array);
}

/**
 * @brief Allocate a 3D CUDA array that is addressable using square brackets
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @param o The third dimension of the array
 * @return double*** A 3D array
 */
void alloc_3d_cuda_array(int m, int n, int o, double **array, size_t *pitch) {
	hipMallocPitch((void **)array, pitch, n*o*sizeof(double), m);
	*pitch = (*pitch) / sizeof(double);
	printf("Pitch is %ld\n", *pitch);
	double *tmp = (double *) calloc((*pitch)*m, sizeof(double));
	hipMemcpy(*array, tmp, (*pitch)*m*sizeof(double), hipMemcpyHostToDevice);
}

/**
 * @brief Free a 3D CUDA array
 * 
 * @param array The 3D array to free
 */
void free_3d_cuda_array(double* array) {
	hipFree(array);
}

/**
 * @brief Allocate a 3D C array that is addressable using square brackets
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @param o The third dimension of the array
 * @return double*** A 3D array
 */
double ***alloc_3d_array(int m, int n, int o) {
	double ***x;
	x = (double***) malloc(m*sizeof(double **));
	x[0] = (double **) malloc(m*n*sizeof(double *));
	x[0][0] = (double *) calloc(m*n*o,sizeof(double));
	for (int i = 1; i < m; i++) {
		x[i] = &x[0][i*n];
	}
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			if (i == 0 && j == 0) continue;
			x[i][j] = &x[0][0][i*n*o + j*o];
		}
	}
	return x;
}

/**
 * @brief Free a 3D C array
 * 
 * @param array The 3D array to free
 */
void free_3d_array(double*** array) {
	free(array[0][0]);
	free(array[0]);
	free(array);
}
